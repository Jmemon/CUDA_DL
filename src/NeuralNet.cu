#include "../include/NeuralNet.h"
#include <stdlib.h>
#include <stdio.h>

NeuralNet::NeuralNet(int *l, Activation *f, int n) {
	
	num_layers = n;
	layers = l;
	funcs = f;

	weights = (double **)malloc((num_layers - 1) * sizeof(double *));
	for (int i = 0; i < num_layers - 1; i++) {
		*(weights + i) = (double *)malloc(layers[i] * layers[i + 1] * sizeof(double));
	}

	initWeights();

}

void NeuralNet::initWeights() {

	int i;
	double** d_w = (double **)malloc((num_layers - 1) * sizeof(double *));

	for (i = 0; i < num_layers - 1; i++) {

		int num_weights = layers[i] * layers[i + 1];
		size_t SIZE = num_weights * sizeof(double);
		
		hipMalloc((void **)&(*(d_w + i)), SIZE);

		hipMemcpy(*(d_w + i), *(weights + i), SIZE, hipMemcpyHostToDevice);

		dim3 BLOCKS(num_weights / 1024 + 1, 1, 1);
		dim3 THREADS(num_weights / (num_weights / 1024 + 1), 1, 1);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);
		randInit<<<BLOCKS, THREADS>>>(*(d_w + i));
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		hipMemcpy(*(weights + i), *(d_w + i), SIZE, hipMemcpyDeviceToHost);

		float ms;
		hipEventElapsedTime(&ms, start, stop);

		printf("W%d Init Time: %f \n", i + 1, ms);
		printf("W%d Size: %d x %d \n\n", i + 1, layers[i + 1], layers[i]);

		hipFree(*(d_w + i));
	}

	free(d_w);

}

void NeuralNet::printWeights(double* w, int l1, int l2) const {
	
	for (int i = 0; i < l2; i++) {
		
		for (int j = 0; j < l1; j++) {
			printf("%f  ", w[i * l1 + j]);
		}
		printf("\n");

	}
	printf("\n");

}

void NeuralNet::activation(double *x, int len, Activation f) {
	
	size_t SIZE = len * sizeof(double)

	double *d_x;
	hipMalloc((void **) &d_x, SIZE);

	hipMemcpy(d_x, x, SIZE, hipMemcpyHostToDevice);

	dim3 BLOCKS(len / 1024 + 1, 1, 1);
	dim3 THREADS(len / (len / 1024 + 1), 1, 1);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	if (f == binary_step) {
		hipEventRecord(start);
		binary_step<<<BLOCKS, THREADS>>>(d_x);
		hipEventRecord(stop);
	} /*else if (f == sigmoid) {
		hipEventRecord(start);
		sigmoid<<<BLOCKS, THREADS>>>(d_x);
		hipEventRecord(stop);
	*/} else if (f == relu) {
		hipEventRecord(start);
		relu<<<BLOCKS, THREADS>>>(d_x);
		hipEventRecord(stop);
	} else if (f == leaky_relu) {
		hipEventRecord(start);
		leaky_relu<<<BLOCKS, THREADS>>>(d_x);
		hipEventRecord(stop);
	} else {
		cout << "Activation function must be binary_step, relu, or leaky_relu" << endl;"
	}

	hipEventSynchronize(stop);

	hipMemcpy(x, d_x, SIZE, hipMemcpyDeviceToHost);

	double ms;
	hipEventElapsedTime(&ms, start, stop);

	cout << "Activation Time: " << ms << endl;
}
