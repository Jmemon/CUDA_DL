#include "hip/hip_runtime.h"
#include "../include/NeuralNet.h"
#include "../include/Activation.cuh"
#include "../include/Matrix.cuh"
#include "../include/Loss.cuh"
#include <iostream>
#include <stdio.h>
#include <exception>
#include <vector>

/* -------------------------------------------------- 
Constructor 

Parameters: 
	l - l[i] is number of neurons in layer i
	f - f[i] is activation func for layer i + 1
	e - loss function

Initializes layers to l, funcs to f, errFunc to e
Randomly initializes weights to values between 0 and 1, 
	We determine the sizes of the weight matrices using the values given in l
-------------------------------------------------- */
NeuralNet::NeuralNet(std::vector<int> &l, std::vector<Activation> &f, Loss e) 
	: layers(l), funcs(f), errFunc(e)
{
	if (l.size() < 2)
		throw std::length_error("Network must at least have input and output layer");
	
	if (f.size() != l.size() - 1)
		throw std::length_error("Every layer must have activation except input");

	for (int i = 0; i < layers.size() - 1; i++) 
	{
		std::vector<double> tmp(l[i] * l[i + 1] , 0);
		
		for (int j = 0; j < tmp.size(); j++)
			tmp[j] = (double)(rand() % 10000) / 10000;

		weights.push_back(tmp);

	} // end for

} // end NeuralNet

/* -------------------------------------------------- 
activation

Parameters: 
	x - vector to apply activation func to 
	f - activation func to apply to x

Uses the cuda kernels defined in Activation.cu on x

Returns:
	a - vector equal to f(x)
-------------------------------------------------- */
std::vector<double> NeuralNet::activation(std::vector<double> &x, Activation f, bool diff) 
{
	if (x.size() < 1)
		throw std::length_error("Layer must have at least one node");

	std::vector<double> a(x.size());

	switch(f)
	{
		case binary_step:

			if (diff)
				throw std::domain_error("activation: No binStep deriv implemented");

			a = binaryStepGPU(x);
			break;
		case sigmoid:
			a = sigmoidGPU(x, diff);
			break;
		case relu:
			a = reluGPU(x, diff);
			break;
		case leaky_relu:
			a = leakyReluGPU(x, diff);
			break;
		default:
			throw std::domain_error("This activation functions is not implemented.");
	} // end switch

	return a;
} // end activation

/* -------------------------------------------------- 
forwardPass

Parameters: 
	x - vector to apply activation func to
	  - can be a matrix in Row-Major form 

Puts input x through the network and generates a prediction
If x is a matrix where each column is one input, it will do all as a batch
Stores each layer's unactivated value and activated one too for output layer

Returns:
	out - vector with the intermediate values and output 
		- if network is n-m-k and batch_size is 5, then out will have
		5m + 5k + 5k elements, 5 because of the batchsize
-------------------------------------------------- */
std::vector<std::vector<double> > NeuralNet::forwardPass(std::vector<double> &x)
{
	// -- Error Check --------------------------------------------------------
	double tmp = (double)(x.size()) / (double)(layers[0]);

	if (tmp < 1.0)
	{
		char msg [100];
		std::sprintf(msg, "User Input Size: %lud ; NN Input Size: %d", x.size(), layers[0]);
		throw std::length_error(msg);
	} // end if
	
	if (std::floor(tmp) != tmp)
	{
		char msg [100];
		std::sprintf(msg, "Too many/few Input Args (in_size / nn_in_size = %f)", tmp);
		throw std::length_error(msg);
	} // end if
	// ----------------------------------------------------------------------

	int batch_size = x.size() / layers[0];  // num cols in x
	int input_size = layers[0];				// num rows in x
	std::vector<double> tmp_v(x);
	std::vector<std::vector<double> > out(layers.size() + 1); 	

	out[0] = x;

	for (int i = 1; i < layers.size(); i++)
	{
		out[i] = matMulGPU(weights[i - 1], tmp_v, layers[i], layers[i - 1], batch_size);
		// out[i] = z_(i + 1)
		// weights[i] is layers[i + 1] x layers[i]
		// tmp_v is layers[i] x batch_size
		// out[i] is layers[i + 1] x batch_size

		tmp_v = activation(out[i], funcs[i - 1]);	
		// tmp_v = a_(i + 1)

	} // end for

	*(out.end() - 1) = tmp_v;

	return out;
} // end forwardPass

/* -------------------------------------------------- 
calcLoss

Parameters: 
	x - vector of predicted outputs 
	  - can be a matrix in Row-Major form 
	y - vector of actual outputs
	  - can be a matrix in Row-Major form

Applies whatever loss function is specified by NeuralNet.errFunc

Returns:
	err - double which is average error for batch of inputs
-------------------------------------------------- */
double NeuralNet::calcLoss(std::vector<double>& x, std::vector<double>& y)
{
	// -- Error Check --------------------------------------------------------
	double tmp1 = (double)(x.size()) / (double)(layers[layers.size() - 1]);
	double tmp2 = (double)(y.size()) / (double)(layers[layers.size() - 1]);

	if (tmp1 < 1.0) 
	{
		char msg [100];
		std::sprintf(msg, "User Output Size: %lud ; NN Output Size: %d", x.size(), layers[layers.size() - 1]);
		throw std::length_error(msg);
	} // end if
	
	if (tmp2 < 1.0) 
	{
		char msg [100];
		std::sprintf(msg, "User Output Size: %lud ; NN Output Size: %d", y.size(), layers[layers.size() - 1]);
		throw std::length_error(msg);
	} // end if

	if (std::floor(tmp1) != tmp1)
	{
		char msg [100];
		std::sprintf(msg, "Too many/few Input Args (out_size / nn_out_size = %f)", tmp1);
		throw std::length_error(msg);
	} // end if

	if (std::floor(tmp2) != tmp2)
	{
		char msg [100];
		std::sprintf(msg, "Too many/few Input Args (out_size / nn_out_size = %f)", tmp2);
		throw std::length_error(msg);
	} // end if
	// ----------------------------------------------------------------------

	double err;

	switch (errFunc)
	{
		case mse:
			err = mseGPU(x, y, x.size() / layers[layers.size() - 1]);
			break;
		case logLoss:
			err = crossEntropyGPU(x, y, x.size() / layers[layers.size() - 1]);
			break;
		default:
			throw std::domain_error("This loss function has not been implemented");
	} // end switch

	return err;
} // end error

/* -------------------------------------------------- 
backwardPass

Parameters: 
	FP - result of forwardPass (should have z's for every layer and output value)
	y - actual output
	batch_size - samples in batch

Uses vector iterators to move back through the network and get the gradients for Cost wrt weights

Returns:
	dC - vector of vectors where each vector is the deriv of cost wrt some weight
-------------------------------------------------- */
std::vector<std::vector<double> > NeuralNet::backwardPass(std::vector<std::vector<double> > &FP, std::vector<double> &y, int batch_size)
{

	// ---Error Check------------------------------------------------
	for (int i = 0; i < layers.size(); i++)
	{
		if (FP[i].size() / batch_size != layers[i])
			throw std::length_error("backwardPass: Invalid Vector size to FP");
		
	} // end for

	if (FP.back().size() / batch_size != layers.back())
		throw std::length_error("backwardsPass: Invalid Vector size to FP");
	
	if (y.size() / batch_size != layers[layers.size() - 1])
		throw std::length_error("backwardPass: Invalid Vector size to y");

	if (batch_size < 1)
		throw std::invalid_argument("backwardPass: there must be at least one sample in batch");
	// -------------------------------------------------------------- 

	std::vector<std::vector<double> > dC(layers.size() - 1);
	std::vector<double> delta, a, w;

	std::vector<std::vector<double> >::iterator it_dC = dC.end() - 1, it_FP = FP.end() - 1, it_weights = weights.end() - 1;
	std::vector<int>::iterator it_layers = layers.end() - 1;
	std::vector<Activation>::iterator it_funcs = funcs.end() - 1;
	
	switch (errFunc)
	{
		case mse:
			delta = msePrimeGPU(*it_FP, y, *it_layers, batch_size);
			break;
		case logLoss:
			delta = crossEntropyPrimeGPU(*it_FP, y, *it_layers, batch_size);
			break;
		default:
			throw std::domain_error("backwardPass: This loss func hasn't been implemented");
	} // end switch

	// a <- act'(zL)
	a = activation(*(it_FP - 1), *it_funcs, true);

	// delta <- del_a(C) o act'(zL) = deltaL
	delta = hadamardGPU(delta, a, *it_layers, batch_size);

	// a <- a(z(L-1))
	a = activation(*(it_FP - 2), *(it_funcs - 1));

	// a <- act(z(L-1))^T
	a = matTransGPU(a, *(it_layers - 1), batch_size);

	// dC[l] <- (deltaL)(a[L-1])^T
	*it_dC = matMulGPU(delta, a, *it_layers, batch_size, *(it_layers - 1));

	// pts to z(L-1)
	it_FP -= 2;

	// pts to layers[L - 1]
	it_layers -= 1;

	// pts to funcs[L - 1]
	it_funcs -= 1;

	// pts to it_dC[L - 1]
	it_dC -= 1;

	// it_weights already points to weights[L]

	while (it_dC != dC.begin())
	{
		// l denotes layer 

		// w <- w[l+1]^T    [l+1 x l --> l x l+1] 
		w = matTransGPU(*it_weights, *(it_layers + 1), *(it_layers));

		// delta <- (w[l+1]^T)(delta[l+1])
		delta = matMulGPU(w, delta, *it_layers, *(it_layers + 1), batch_size);

		// a <- act'(z[l])
		a = activation(*it_FP, *it_funcs, true);

		// delta <- delta[l]
		delta = hadamardGPU(delta, a, *it_layers, batch_size);

		// a <- act(z(l-1))
		a = activation(*(it_FP - 1), *(it_funcs - 1));

		// a <- act(z(l-1))^T
		a = matTransGPU(a, *(it_layers - 1), batch_size);

		// it_dC[l] <- (delta[l])(act[l-1]^T)
		*it_dC = matMulGPU(delta, a, *it_layers, batch_size, *(it_layers - 1));

		// decrement iterators
		it_FP -= 1;
		it_layers -= 1;
		it_funcs -= 1;
		it_weights -= 1;
		it_dC -= 1;	
	} // end while

	// w <- w[l+1]^T    [l+1 x l --> l x l+1] 
	w = matTransGPU(*it_weights, *(it_layers + 1), *(it_layers));

	// delta <- (w[l+1]^T)(delta[l+1])
	delta = matMulGPU(w, delta, *it_layers, *(it_layers + 1), batch_size);

	// a <- act'(z2)
	a = activation(*it_FP, *it_funcs, true);

	// delta <- delta2
	delta = hadamardGPU(delta, a, *it_layers, batch_size);

	// a <- x
	a = *(it_FP - 1);

	// a <- x^T
	a = matTransGPU(a, *(it_layers - 1), batch_size);

	// it_dC[1] <- (delta[1])(x^T)
	*it_dC = matMulGPU(delta, a, *it_layers, batch_size, *(it_layers - 1));

	return dC;
} // end backwardPass

/* -------------------------------------------------- 
printNN

Prints the size of each layer, the activation function
	at each layer, and the loss function at the end of the network
-------------------------------------------------- */
void NeuralNet::printNN() const
{
	std::cout << "Layer 0: " << layers[0] << std::endl;	

	for (int i = 1; i < layers.size(); i++)
	{
		std::cout << std::endl;
		std::cout << "Layer " << i << ": " << layers[i] << std::endl;
		
		switch (funcs[i - 1])
		{
			case binary_step: 
				std::cout << "Activation: Binary Step" << std::endl;
				break;
			case sigmoid: 
				std::cout << "Activation: Sigmoid" << std::endl;
				break;
			case relu: 
				std::cout << "Activation: ReLU" << std::endl;
				break;
			case leaky_relu: 
				std::cout << "Activation: Leaky ReLU" << std::endl;
				break;
			default:
				throw std::domain_error("This activation function is not implemented");
		} // end switch

	} // end for

	std::cout << std::endl;

	switch(errFunc)
	{
		case mse:
			std::cout << "Loss Function: Mean-Squared Error" << std::endl;
			break;
		case logLoss:
			std::cout << "Loss Function: Cross Entropy" << std::endl;
			break;
		default:
			throw std::domain_error("This loss function is not implemented");
	} // end switch

	std::cout << std::endl;

} // end printNN

/* -------------------------------------------------- 
printWeights

Parameter:
	l - layers to print weights for (range is 0 to layers.size() - 2)

Prints all the weights from layer l to l + 1
-------------------------------------------------- */
void NeuralNet::printWeights(int l) const 
{
	if (l < 0)
		throw std::length_error("Not a layer");

	if (l > layers.size() - 2)
		throw std::domain_error("There are no weights for this layer");

	std::cout << "Weights for layers " << l << " to " << l + 1 << ":" << std::endl;
	
	for (int i = 0; i < layers[l + 1]; i++) 
	{	
		for (int j = 0; j < layers[l]; j++) 
			std::cout << weights[l][i * layers[l] + j] << "  ";			
		
		std::cout << std::endl;

	} // end for

	std::cout << std::endl;

} // end printWeights

