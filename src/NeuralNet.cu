#include "../include/NeuralNet.h"
#include "../include/Activation.cuh"
#include <vector>
#include <iostream>

NeuralNet::NeuralNet(const std::vector<int> l, const std::vector<Activation> f) 
	: num_layers(l.size()), layers(l), funcs(f)
{
	
	for (int i = 0; i < num_layers - 1; i++) 
	{
		std::vector<double> tmp(l[i] * l[i + 1] , 0);
		
		for (int j = 0; j < tmp.size(); j++)
			tmp[j] = (double)(rand() % 10000) / 10000;

		weights.push_back(tmp);

	} // end for

} // end NeuralNet

void NeuralNet::printWeights(int l) const 
{

	if (l >= num_layers - 1)
	{
		std::cout << "l is too large" << std::endl;
		return;
	}

	std::cout << "Weights for layers " << l << " to " << l + 1 << ":" << std::endl;
	
	for (int i = 0; i < layers[l + 1]; i++) 
	{	
		for (int j = 0; j < layers[l]; j++) 
			std::cout << weights[l][i * layers[l] + j] << "  ";			
		
		std::cout << std::endl;

	} // end for

	std::cout << std::endl;

} // end printWeights

void NeuralNet::activation(std::vector<double> x, Activation f) 
{	
	size_t SIZE = x.size() * sizeof(double);

	double *d_x;
	hipMalloc((void **) &d_x, SIZE);

	hipMemcpy(d_x, x.data(), SIZE, hipMemcpyHostToDevice);

	dim3 BLOCKS(x.size() / 1024 + 1, 1, 1);
	dim3 THREADS(x.size() / (x.size() / 1024 + 1), 1, 1);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	if (f == binary_step) {
		hipEventRecord(start);
		binaryStepGPU(d_x, BLOCKS, THREADS);	
		hipEventRecord(stop);
	} /*else if (f == sigmoid) {
		hipEventRecord(start);
		sigmoidGPU(d_x, BLOCKS, THREADS);
		hipEventRecord(stop);
	}*/ else if (f == relu) {
		hipEventRecord(start);
		reluGPU(d_x, BLOCKS, THREADS);
		hipEventRecord(stop);
	} else if (f == leaky_relu) {
		hipEventRecord(start);
		leakyReluGPU(d_x, BLOCKS, THREADS);
		hipEventRecord(stop);
	} else {
		std::cout << "Activation function must be binary_step, relu, or leaky_relu" << std::endl;
	}

	hipEventSynchronize(stop);

	hipMemcpy(x.data(), d_x, SIZE, hipMemcpyDeviceToHost);

	float ms;
	hipEventElapsedTime(&ms, start, stop);

	std::cout << "Activation Time: " << ms << std::endl;

} // end activation
