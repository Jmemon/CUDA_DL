#include "../include/NeuralNet.h"
#include "../include/Activation.cuh"
#include <vector>
#include <iostream>

NeuralNet::NeuralNet(const vector<int> l, const vector<Activation> f) 
	: num_layers(l.size()), layers(l), funcs(f)
{
	
	for (int i = 0; i < num_layers - 1; i++) 
	{
		vector<double> tmp(l[i] * l[i + 1] , 0);
		
		for (int j = 0; j < tmp.size(); j++)
			tmp[j] = (double)(rand() % 10000) / 10000;

		weights.push_back(tmp);

	} // end for

} // end NeuralNet

/* void NeuralNet::initWeights() {

	int i;
	double** d_w = (double **)malloc((num_layers - 1) * sizeof(double *));

	for (i = 0; i < num_layers - 1; i++) {

		int num_weights = layers[i] * layers[i + 1];
		size_t SIZE = num_weights * sizeof(double);
		
		hipMalloc((void **)&(*(d_w + i)), SIZE);

		hipMemcpy(*(d_w + i), *(weights + i), SIZE, hipMemcpyHostToDevice);

		dim3 BLOCKS(num_weights / 1024 + 1, 1, 1);
		dim3 THREADS(num_weights / (num_weights / 1024 + 1), 1, 1);

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		hipEventRecord(start);
		randInitGPU(*(d_w + i), BLOCKS, THREADS);
		hipEventRecord(stop);

		hipEventSynchronize(stop);

		hipMemcpy(*(weights + i), *(d_w + i), SIZE, hipMemcpyDeviceToHost);

		float ms;
		hipEventElapsedTime(&ms, start, stop);

		printf("W%d Init Time: %f \n", i + 1, ms);
		printf("W%d Size: %d x %d \n\n", i + 1, layers[i + 1], layers[i]);

		hipFree(*(d_w + i));
	}

	free(d_w);

} */

void NeuralNet::printWeights(vector<double> w, int l1, int l2) const 
{	
	for (int i = 0; i < l2; i++) 
	{	
		for (int j = 0; j < l1; j++) 
			std::cout << w[i * l1 + j] << "  ";			
		
		std::cout << std::endl;

	} // end for

	std::cout << std::endl;

} // end printWeights

void NeuralNet::activation(vector<double> x, Activation f) 
{	
	size_t SIZE = x.size() * sizeof(double);

	double *d_x;
	hipMalloc((void **) &d_x, SIZE);

	hipMemcpy(d_x, x.data(), SIZE, hipMemcpyHostToDevice);

	dim3 BLOCKS(len / 1024 + 1, 1, 1);
	dim3 THREADS(len / (len / 1024 + 1), 1, 1);
	
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	if (f == binary_step) {
		hipEventRecord(start);
		binaryStepGPU(d_x, BLOCKS, THREADS);	
		hipEventRecord(stop);
	} /*else if (f == sigmoid) {
		hipEventRecord(start);
		sigmoid<<<BLOCKS, THREADS>>>(d_x);
		hipEventRecord(stop);
	}*/ else if (f == relu) {
		hipEventRecord(start);
		reluGPU(d_x, BLOCKS, THREADS);
		hipEventRecord(stop);
	} else if (f == leaky_relu) {
		hipEventRecord(start);
		leakyReluGPU(d_x, BLOCKS, THREADS);
		hipEventRecord(stop);
	} else {
		std::cout << "Activation function must be binary_step, relu, or leaky_relu" << std::endl;
	}

	hipEventSynchronize(stop);

	hipMemcpy(x.data(), d_x, SIZE, hipMemcpyDeviceToHost);

	float ms;
	hipEventElapsedTime(&ms, start, stop);

	std::cout << "Activation Time: " << ms << std::endl;

} // end activation
