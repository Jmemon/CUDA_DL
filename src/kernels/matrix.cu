#include "hip/hip_runtime.h"
#include "../../include/kernels/matrix.cuh"
#include <vector>
#include <algorithm> // std::max

/* ---------------------------------------------------------------
affineTransformGPU

Parameters: 
	x - double ptr representing matrix X in row-major form
	weights - double ptr representing matrix W in row-major form
	bias - double ptr representing matrix B in row-major form
	m - rows in X
	n - cols in X
	k - cols in W

Performs affine transformation: XW^T + B
--------------------------------------------------------------- */
__global__ void affineTransform(double *x, double *weights, double *bias, double *output, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y;
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0.0;
	
	if (row >= m || col >= k)
		return;
	
	if (col < k && row < m)
	{
		for (int i = 0; i < n; i++)
			sum += x[row * n + i] * weights[i * k + col];
	}
	
	output[row * k + col] = sum + bias[col];
} // end affineTransform

/* ---------------------------------------------------------------
affineTransformGPU

Parameters: 
	x - vector representing matrix X
	weights - vector representing matrix W
	bias - vector representing matrix B
	m - rows in X
	n - cols in X
	k - cols in W

Calls cuda kernel affineTransform on x.data(), weights.data(), and bias.data()

Returns:
	output - vector representing XW^T + B (has dim m x k)
--------------------------------------------------------------- */
std::vector<double> affineTransformGPU(std::vector<double>& x, std::vector<double>& weights, std::vector<double>& bias, int m, int n, int k)
{
	double *d_x, *d_weights, *d_bias, *d_output;
	std::vector<double> output(m * k);
	int BLOCKSIZE = m >= 32 || k >= 32 ? 32 : std::max(m, k);
	
	size_t SIZE = m * n * sizeof(double);
	
	hipMalloc((void **) &d_x, SIZE);
	hipMalloc((void **) &d_weights, SIZE);
	hipMalloc((void **) &d_bias, k * sizeof(double));
	hipMalloc((void **) &d_output, m * k * sizeof(double));

	hipMemcpy(d_x, x.data(), SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_weights, weights.data(), SIZE, hipMemcpyHostToDevice);
	hipMemcpy(d_bias, bias.data(), k * sizeof(double), hipMemcpyHostToDevice);

	dim3 GRID((k + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	affineTransform<<<GRID, BLOCK, 0>>>(d_x, d_weights, d_bias, d_output, m, n, k);
	hipDeviceSynchronize();

	hipMemcpy(output.data(), d_output, m * k * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_x);
	hipFree(d_weights);
	hipFree(d_bias);
	hipFree(d_output);

	return output;
} // end affineTransformGPU


/* ---------------------------------------------------------------
matMul

Parameters: 
	a - double ptr representing matrix A in row-major form
    b - double ptr representing matrix B in row-major form
    c - double ptr where AB will be stored in row-major form
	m - rows in A / C
	n - cols in A / rows in B
	k - cols in B / C

Multiplies the matrices stored in row-major form in a and b, then stores
	the output in c

Could be optimized much further with shared memory
--------------------------------------------------------------- */
__global__ void matMul(double *a, double *b, double *c, int m, int n, int k)
{
	int row = blockIdx.y * blockDim.y + threadIdx.y; 
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	double sum = 0.0;
   
	if (row >= m || col >= k)
		return;

	if (col < k && row < m) 
	{
		for(int i = 0; i < n; i++) 
			sum += a[row * n + i] * b[i * k + col];
        
		c[row * k + col] = sum;
	} // end if

} // end matMul

/* ---------------------------------------------------------------
matMulGPU

Parameters: 
	a - vector representing first matrix
    b - vector representing second matrix
	m - rows in a
	n - cols in a / rows in b
	k - cols in b

Calls cuda kernel matMul on a.data() and b.data()

Returns:
	c - vector representing AB (has dim m x k)
--------------------------------------------------------------- */
std::vector<double> matMulGPU(std::vector<double>& a, std::vector<double>& b, int m, int n, int k)
{
	double *d_a, *d_b, *d_c;
	std::vector<double> c(m * k);
	int BLOCKSIZE = m >= 32 || k >= 32 ? 32 : std::max(m, k);	

	hipMalloc((void **) &d_a, m * n * sizeof(double));
	hipMalloc((void **) &d_b, n * k * sizeof(double));
	hipMalloc((void **) &d_c, m * k * sizeof(double));

	hipMemcpy(d_a, a.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), n * k * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 GRID((k + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	matMul<<<GRID, BLOCK, 0>>>(d_a, d_b, d_c, m, n, k);
	hipDeviceSynchronize();

	hipMemcpy(c.data(), d_c, m * k * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return c;
} // end matMulGPU

/* ---------------------------------------------------------------
scalarMult

Parameters: 
	a - double ptr representing matrix A in row-major form
	c - scalar to multiply a by
	len - int representing length of row-major representation of A

Performs scalar multiplication and stores result in a
--------------------------------------------------------------- */
__global__ void scalarMult(double *a, double c, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;

	if (g_idx >= len)
		return;

	a[g_idx] = c * a[g_idx];
} // end scalarMult

/* ---------------------------------------------------------------
scalarMultGPU

Parameters: 
	a - vector representing matrix A
	c - scalar to multiply a by
	m - rows in A 
	n - cols in A 

Calls cuda kernel scalarMult on a.data()

Returns:
	B - vector representing cA (has dim m x n)
--------------------------------------------------------------- */
std::vector<double> scalarMultGPU(std::vector<double>& a, double c, int m, int n)
{
	double *d_a;
	std::vector<double> b(m * n);	
	int BLOCKSIZE = m >= 32 || n >= 32 ? 32 : std::max(m, n);

	hipMalloc((void **) &d_a, m * n * sizeof(double));

	hipMemcpy(d_a, a.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 GRID((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	scalarMult<<<GRID, BLOCK, 0>>>(d_a, c, m * n);
	hipDeviceSynchronize();

	hipMemcpy(b.data(), d_a, m * n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_a);

	return b;
} // end scalarMultGPU

/* ---------------------------------------------------------------
hadamard

Parameters: 
	a - double ptr representing matrix A in row-major form
    b - double ptr representing matrix B in row-major form
    c - double ptr where A o B will be stored in row-major form
	len - the length of row-major form of A, B, and C

Performs Hadamard operation (element-wise mult) and stores result in c
--------------------------------------------------------------- */
__global__ void hadamard(double *a, double *b, double *c, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockIdx.y * blockDim.y + threadIdx.y)
		+ blockIdx.x * blockDim.x + threadIdx.x;

	if (g_idx >= len)
		return;

	c[g_idx] = a[g_idx] + b[g_idx];
} // end haramard

/* ---------------------------------------------------------------
hadamardGPU

Parameters: 
	a - vector representing matrix A
    b - vector representing matrix B
	m - rows in A / B
	n - cols in A / B

Calls cuda kernel hadamard on a.data() and b.data()

Returns:
	c - vector representing A o B (has dim m x n)
--------------------------------------------------------------- */
std::vector<double> hadamardGPU(std::vector<double>& a, std::vector<double>& b, int m, int n)
{
	double *d_a, *d_b, *d_c;
	std::vector<double> c(m * n);	
	int BLOCKSIZE = m >= 32 || n >= 32 ? 32 : std::max(m, n);

	hipMalloc((void **) &d_a, m * n * sizeof(double));
	hipMalloc((void **) &d_b, m * n * sizeof(double));
	hipMalloc((void **) &d_c, m * n * sizeof(double));

	hipMemcpy(d_a, a.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 GRID((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	hadamard<<<GRID, BLOCK, 0>>>(d_a, d_b, d_c, m * n);
	hipDeviceSynchronize();

	hipMemcpy(c.data(), d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return c;
} // end hadamardGPU

/* ---------------------------------------------------------------
matAdd

Parameters: 
	a - double ptr representing matrix A in row-major form
    b - double ptr representing matrix B in row-major form
    c - double ptr where A + B will be stored in row-major form
	len - the length of row-major form of A, B, and C

Performs A + B and stores result in c
--------------------------------------------------------------- */
__global__ void matAdd(double *a, double *b, double *c, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockIdx.y * blockDim.y + threadIdx.y)
		+ blockIdx.x * blockDim.x + threadIdx.x;
	
	if (g_idx >= len)
		return;

	c[g_idx] = a[g_idx] + b[g_idx];
} // end matAdd

/* ---------------------------------------------------------------
matAddGPU

Parameters: 
	a - vector representing matrix A
    b - vector representing matrix B
	m - rows in A / B
	n - cols in A / B

Calls cuda kernel matAdd on a.data() and b.data()

Returns:
	c - vector representing A + B (has dim m x n)
--------------------------------------------------------------- */
std::vector<double> matAddGPU(std::vector<double>& a, std::vector<double>& b, int m, int n)
{
	double *d_a, *d_b, *d_c;
	std::vector<double> c(m * n);	
	int BLOCKSIZE = m >= 32 || n >= 32 ? 32 : std::max(m, n);

	hipMalloc((void **) &d_a, m * n * sizeof(double));
	hipMalloc((void **) &d_b, m * n * sizeof(double));
	hipMalloc((void **) &d_c, m * n * sizeof(double));

	hipMemcpy(d_a, a.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 GRID((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	matAdd<<<GRID, BLOCK, 0>>>(d_a, d_b, d_c, m * n);
	hipDeviceSynchronize();

	hipMemcpy(c.data(), d_c, m * n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	
	return c;
} // end matAddGPU

/* ---------------------------------------------------------------
matReciprocal

Parameters: 
	a - double ptr representing matrix A in row-major form
	len - length of vector representing A

raises each value in A to the -1 power
--------------------------------------------------------------- */
__global__ void matReciprocal(double *a, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;
	
	if (len >= g_idx)
		return;

	a[g_idx] = 1.0 / a[g_idx];
} // end matReciprocal

/* ---------------------------------------------------------------
matReciprocalGPU

Parameters: 
	a - vector representing matrix A
	m - rows in matrix A
	n - cols in matrix A

Calls cuda kernel matReciprocal on a.data()

Returns:
	c - vector representing reciprocal A 
--------------------------------------------------------------- */
std::vector<double> matReciprocalGPU(std::vector<double>& a, int m, int n)
{
	double *d_a;
	std::vector<double> c(m * n);
	int BLOCKSIZE = m >= 32 || n >= 32 ? 32 : std::max(m, n);	

	hipMalloc((void **) &d_a, m * n * sizeof(double));

	hipMemcpy(d_a, a.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 GRID((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	matReciprocal<<<GRID, BLOCK, 0>>>(d_a, m * n); 
	hipDeviceSynchronize();

	hipMemcpy(c.data(), d_a, m * n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_a);
	
	return c;
} // end matMulGPU

/* ---------------------------------------------------------------
matSqrt

Parameters: 
	a - double ptr representing matrix A in row-major form
	len - length of vector representing A

Square root of each value in A
--------------------------------------------------------------- */
__global__ void matSqrt(double *a, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;
	
	if (len >= g_idx)
		return;

	a[g_idx] = sqrt(a[g_idx]);
} // end matSqrt

/* ---------------------------------------------------------------
matSqrtGPU

Parameters: 
	a - vector representing matrix A
	m - rows in matrix A
	n - cols in matrix A

Calls cuda kernel matSqrt on a.data()

Returns:
	c - vector representing sqrt A 
--------------------------------------------------------------- */
std::vector<double> matSqrtGPU(std::vector<double>& a, int m, int n)
{
	double *d_a;
	std::vector<double> c(m * n);
	int BLOCKSIZE = m >= 32 || n >= 32 ? 32 : std::max(m, n);	

	hipMalloc((void **) &d_a, m * n * sizeof(double));

	hipMemcpy(d_a, a.data(), m * n * sizeof(double), hipMemcpyHostToDevice);
	
	dim3 GRID((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	matSqrt<<<GRID, BLOCK, 0>>>(d_a, m * n); 
	hipDeviceSynchronize();

	hipMemcpy(c.data(), d_a, m * n * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_a);
	
	return c;
} // end matSqrtGPU

/* ---------------------------------------------------------------
matTrans

Parameters: 
	a - double ptr representing matrix A in row-major form
    aT - double ptr representing matrix AT in row-major form
	m - rows in A / cols in AT
	n - cols in A / rows in AT

Transposes matrix A
--------------------------------------------------------------- */
__global__ void matTrans(double *a, double *aT, int m, int n)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;
	int row = blockIdx.y * blockDim.y + threadIdx.y;

	if (col >= n || row >= m)
		return;

	if (col < n && row < m) 
	{
		int pos = row * n + col;
		int trans_pos = col * m + row;
		aT[trans_pos] = a[pos];
	} // end if

} // end matTrans

/* ---------------------------------------------------------------
matTransGPU

Parameters: 
	a - vector representing matrix A
	m - rows in A / cols in AT
	n - cols in A / rows in AT

Calls cuda kernel matTrans on a.data()

Returns:
	aT - vector representing AT
--------------------------------------------------------------- */
std::vector<double> matTransGPU(std::vector<double>& a, int m, int n)
{
	double *d_a, *d_aT;
	std::vector<double> aT(m * n);
	int BLOCKSIZE = m >= 32 || n >= 32 ? 32 : std::max(m, n);

	size_t SIZE = m * n * sizeof(double);

	hipMalloc((void **) &d_a, SIZE); 
	hipMalloc((void **) &d_aT, SIZE);

	hipMemcpy(d_a, a.data(), SIZE, hipMemcpyHostToDevice);

	dim3 GRID((n + BLOCKSIZE - 1) / BLOCKSIZE, (m + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE, BLOCKSIZE);

	matTrans<<<GRID, BLOCK, 0>>>(d_a, d_aT, m, n);
	hipDeviceSynchronize();

	hipMemcpy(aT.data(), d_aT, SIZE, hipMemcpyDeviceToHost);

	hipFree(d_a);
	hipFree(d_aT);

	return aT;
} // end matTransGPU
