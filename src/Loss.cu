#include "hip/hip_runtime.h"
#include "../include/Loss.cuh"
#include <vector>
#include <thrust/transform.h>
#include <thrust/device_vector.h>
#include <iostream>
#include <stdlib.h>

/* ----------------------------------------------
mse_functor

Defines operator that returns squared difference between x and y
For use with thrust::transform()
---------------------------------------------- */
struct mse_functor
{

	mse_functor() {}

	__host__ __device__ double operator() (const double &x, const double &y) const
	{
		return (x - y) * (x - y);
	} // end operator

}; // end mse_functor

/* ----------------------------------------------
mseGPU

Parameters:
	x - vector of predictions, can be matrix in row-major form 
	y - vector of actual outputs, can be matrix in row-major form
	batch_size - number of samples in batch

Uses thrust device_vector and transform to get error
Uses operation defined in mse_functor with thrust::transform
Outputs (1/(2*batch_size))(sum{norm{y-aL}^2})

Returns:
	mse - averaged mean-square error over all samples in batch
---------------------------------------------- */
double mseGPU(std::vector<double> &x, std::vector<double> &y, int batch_size)
{
	double mse = 0.0;
	thrust::device_vector<double> d_x(x);
	thrust::device_vector<double> d_y(y);

	// y[i] <- (x[i] - y[i]) * (x[i] - y[i])
	thrust::transform(d_x.begin(), d_x.end(), d_y.begin(), d_y.begin(), mse_functor());
	
	// set mse to be the sum of squares of each element of every sample of d_y
	mse = thrust::reduce(d_y.begin(), d_y.end(), (double) 0.0, thrust::plus<double>());

	// take batch mse average
	mse /= (2 * batch_size);

	return mse;
} // end mseGPU

/* ----------------------------------------------
msePrime

Parameters:
	x - vector representing predicted output, can be matrix in row-major form
	y - vector representing actual output, can be matrix in row-major form
	len - length of x and y

Stores the derivative of mse in y
Derivative for element i: x_i - y_i
---------------------------------------------- */
__global__ void msePrime(double *x, double *y, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;

	if (g_idx >= len)
		return;

	y[g_idx] = x[g_idx] - y[g_idx];
} // end msePrime

/* ----------------------------------------------
msePrimeGPU

Parameters:
	a - vector of predicted outputs, can be matrix in row-major form
	y - vector of actual outputs, can be matrix in row-major form
	size - number of rows in a/y
	batch_size - number of cols in a/y (number of samples in batch)

Calls msePrime cuda kernel on a and y

Returns:
	dC - vector of derivative of MSE wrt each aL_i
---------------------------------------------- */
std::vector<double> msePrimeGPU(std::vector<double> &a, std::vector<double> &y, int size, int batch_size)
{
	double *d_a, *d_y;
	std::vector<double> dC(a.size());
	int BLOCKSIZE = a.size() >= 512 ? 512 : a.size();

	hipMalloc((void **) &d_a, a.size() * sizeof(double));
	hipMalloc((void **) &d_y, y.size() * sizeof(double));

	hipMemcpy(d_a, a.data(), a.size() * sizeof(double), hipMemcpyHostToDevice);

	dim3 GRID((a.size() + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE);

	msePrime<<<GRID, BLOCK, 0>>>(d_a, d_y, size * batch_size);
	hipDeviceSynchronize();

	hipMemcpy(dC.data(), d_y, a.size() * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < dC.size(); i++)
		dC[i] /= batch_size;

	hipFree(d_a);
	hipFree(d_y);

	return dC;
} // end msePrimeGPU

/* ----------------------------------------------
logLoss_functor

Defines operator that returns -y*log_e(x)
For use with thrust::transform()
---------------------------------------------- */
struct logLoss_functor
{
	
	logLoss_functor() {}

	__host__ __device__ double operator() (const double &x, const double &y) const 
	{
		return -1 * y * log(x);
	}

}; // end ln_functor

/* ----------------------------------------------
crossEntropyGPU

Parameters:
	x - vector of predictions, can be matrix in row-major form 
	y - vector of actual outputs, can be matrix in row-major form
	batch_size - number of samples in batch

Uses thrust device_vector and transform to get error
Uses operation defined in logLoss_functor with thrust::transform
Outputs (1/batch_size)(sum{-y*log_e(x)})

Returns:
	logLoss - cross entropy loss over all samples averaged with batch_size
---------------------------------------------- */
double crossEntropyGPU(std::vector<double> &x, std::vector<double> &y, int batch_size)
{
	double logLoss = 0.0;
	thrust::device_vector<double> d_x(x);
	thrust::device_vector<double> d_y(y);

	// y[i] <- y[i] * log_e(x[i])
	thrust::transform(d_x.begin(), d_x.end(), d_y.begin(), d_y.begin(), logLoss_functor());

	// logLoss <- sum{y[i] * log_e(x[i])}
	logLoss = thrust::reduce(d_y.begin(), d_y.end(), (double) 0.0, thrust::plus<double>());

	// logLoss <- (1/batch_size) * logLoss
	logLoss /= batch_size;

	return logLoss;
} // end crossEntropyGPU

/* ----------------------------------------------
crossEntropyPrime

Parameters:
	a - vector representing predicted output, can be matrix in row-major form
	y - vector representing actual output, can be matrix in row-major form
	len - length of x and y

Stores the derivative of crossEntropy in y
Derivative for element i: -y_i / a_i
---------------------------------------------- */
__global__ void crossEntropyPrime(double *a, double *y, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockIdx.y * blockDim.y + threadIdx.y)
		+ blockIdx.x * blockDim.x + threadIdx.x;
	
	if (g_idx >= len)
		return;

	y[g_idx] = -1 * y[g_idx] / a[g_idx];
} // end crossEntropyPrime

/* ----------------------------------------------
crossEntropyPrimeGPU

Parameters:
	a - vector of predicted outputs, can be matrix in row-major form
	y - vector of actual outputs, can be matrix in row-major form
	size - number of rows in a/y
	batch_size - number of cols in a/y (number of samples in batch)

Calls crossEntropyPrime cuda kernel on a and y

Returns:
	dC - vector of derivative of Cross Entropy wrt each aL_i
---------------------------------------------- */
std::vector<double> crossEntropyPrimeGPU(std::vector<double> &a, std::vector<double> &y, int size, int batch_size)
{
	double *d_a, *d_y;
	std::vector<double> dC(a.size());
	int BLOCKSIZE = a.size() >= 512 ? 512 : a.size();

	hipMalloc((void **) &d_a, a.size() * sizeof(double));
	hipMalloc((void **) &d_y, y.size() * sizeof(double));

	hipMemcpy(d_a, a.data(), a.size() * sizeof(double), hipMemcpyHostToDevice);

	dim3 GRID((a.size() + BLOCKSIZE - 1) / BLOCKSIZE);
	dim3 BLOCK(BLOCKSIZE);

	crossEntropyPrime<<<GRID, BLOCK, 0>>>(d_a, d_y, size * batch_size);
	hipDeviceSynchronize();

	hipMemcpy(dC.data(), d_y, a.size() * sizeof(double), hipMemcpyDeviceToHost);

	for (int i = 0; i < dC.size(); i++)
		dC[i] /= batch_size;

	hipFree(d_a);
	hipFree(d_y);

	return dC;
} // end msePrimeGPU

/*
// block size will always be 512 in this file

__global__ void reduce(double out, double *in, int len)
{
	extern __shared__ double values[];

	double *tmp = new double[gridDim.x * gridDim.y];

	int idx = blockDim.x * threadIdx.y + threadIdx.x;

	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;

	// abort if thread is extra in block
	if (g_idx >= len)
		return;

	// load shared memory array from global memory
	values[idx] = in[g_idx];

	// takes upper half of array and adds it to corresponding elements in lower half
	// does this repeatedly until relevant part of array is one elem
	// 	therefore there are no halves to add
	for (unsigned int i = blockDim.x * blockDim.y / 2; i > 0; i >>= 1)
	{
		if (idx < i)
			values[idx] += values[idx + i];

		__syncthreads();
	} // end for

	// write sum of this portion of array to corresponding index in out
	// all threads will be writing the same value, so we don't need to 
	//	worry about a race condition
	tmp[blockIdx.y * gridDim.x + blockIdx.x] = values[0];

	// if more than one block was used, meaning each block reduced part of array,
	// 	we must reduce further, so call again 
	if (gridDim.x * gridDim.y > 1)
		reduce<<<dim3(len / 512 + 1), dim3(512), (len / 512 + 1) * sizeof(double)>>>(out, tmp, len);
	else
		out = tmp[0];

} // end reduce

// extern means reduce declared elsewhere
// for this application that means its len will be determined based on how much
// 	shared memory we give each block
__global__ void squares(double *sqr, double *x, double *y, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;
	// first line finds number of threads in rows of grid above thread's row 
	// second line finds how far into row thread is
	// so it puts matrix of threads into row major form
	// note that if we have 1d vect of threads, this reduces to threadIdx.x

	// abort if extra thread in block
	if (g_idx >= len)
		return;

	// get array of square differences
	sqr[g_idx] = (x[g_idx] - y[g_idx]) * (x[g_idx] - y[g_idx]);

} // end mse

std::vector<double> mseGPU(double *x, double *y, int size, int e_size) 
{
	double *tmp;
	double *d_err, *d_sqr, *d_x, *d_y;
	std::vector<double> err(e_size);
	thrust::device_vector<double> sqr(size * e_size);

	hipMalloc((void **) &d_err, e_size * sizeof(double));
	hipMalloc((void **) &d_sqr, size * e_size * sizeof(double));
	hipMalloc((void **) &d_x, size * e_size * sizeof(double));
	hipMalloc((void **) &d_y, size * e_size * sizeof(double));

	hipMemcpy(d_x, x, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size * sizeof(double), hipMemcpyHostToDevice);

	dim3 sqrGRID(size * e_size / 512 + 1);
	dim3 sqrBLOCK(512);

	//dim3 redGRID(size / 512 + 1);
	//dim3 redBLOCK(512);

	squares<<<sqrGRID, sqrBLOCK, 0>>>(d_sqr, d_x, d_y, size * e_size);
	
	for (int i = 0; i < e_size; i++)
	{
		tmp = d_sqr + i * size; // it will move through each samples in sqr
		reduce<<<redGRID, redBLOCK, size / redGRID.x * sizeof(double)>>>(d_err[i], tmp, size);
		hipDeviceSynchronize();
	} // end for

	hipMemcpy(err.data(), d_err, e_size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_err);
	hipFree(d_sqr);
	hipFree(d_x);
	hipFree(d_y);

	return err;
} // end mseGPU

__global__ void crossEntropyTerm(double *logLoss, double *x, double *y, int len)
{
	int g_idx = gridDim.x * blockDim.x * (blockDim.y * blockIdx.y + threadIdx.y)
		+ blockDim.x * blockIdx.x + threadIdx.x;
	// first line finds number of threads in rows of grid above thread's row 
	// second line finds how far into row thread is
	// so it puts matrix of threads into row major form
	// note that if we have 1d vect of threads, this reduces to threadIdx.x

	// abort if extra thread in block
	if (g_idx >= len)
		return;

	// get array of log losses
	logLoss[g_idx] = y[g_idx] * log(x[g_idx]);

} // end crossEntropy

std::vector<double> crossEntropyGPU(double *x, double *y, int size, int e_size)
{
	double *tmp;
	double *d_err, *d_term, *d_x, *d_y;
	std::vector<double> err(e_size);

	hipMalloc((void **) &d_err, e_size * sizeof(double));
	hipMalloc((void **) &d_term, size * e_size * sizeof(double));
	hipMalloc((void **) &d_x, size * e_size * sizeof(double));
	hipMalloc((void **) &d_y, size * e_size * sizeof(double));

	hipMemcpy(d_x, x, size * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_y, y, size * sizeof(double), hipMemcpyHostToDevice);

	dim3 trmGRID(size * e_size / 512 + 1);
	dim3 trmBLOCK(512);

	dim3 redGRID(size / 512 + 1);
	dim3 redBLOCK(512);

	crossEntropyTerm<<<trmGRID, trmBLOCK, 0>>>(d_term, d_x, d_y, size * e_size);
	
	for (int i = 0; i < e_size; i++)
	{
		tmp = d_term + i * size; // it will move through each samples in sqr
		reduce<<<redGRID, redBLOCK, size / redGRID.x>>>(d_err[i], tmp, size);
		hipDeviceSynchronize();
	} // end for

	hipMemcpy(err.data(), d_err, e_size * sizeof(double), hipMemcpyDeviceToHost);

	hipFree(d_err);
	hipFree(d_term);
	hipFree(d_x);
	hipFree(d_y);

	return err;
}
*/
